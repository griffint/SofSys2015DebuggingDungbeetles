#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <math.h>
#include <string>
#include <iostream>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/cudaarithm.hpp"

//#define CLUSTER_NUM       20
//#define ITERATION         10

using namespace cv;
using namespace std;
using namespace cuda;

int ROWS, COLS, CLUSTER_NUM, CLUSTER_MAX, ITERATION;

__global__ void GPU_mapToCluster(unsigned char * d_label, Vec3b* d_centroids, uchar* d_image_B, uchar* d_image_G, uchar* d_image_R, int ROWS, int COLS , int CLUSTER_NUM) {
    int col = blockIdx.x;
    int row = threadIdx.x;
    int index = COLS * row + col;
    float temp, a, b, c;
    float d = FLT_MAX;
    unsigned char cluster;
    unsigned char p1_b = d_image_B[index];
    unsigned char p1_g = d_image_G[index];
    unsigned char p1_r = d_image_R[index];
    for (unsigned char i = 0; i < CLUSTER_NUM; i++) {
        a = (float) (p1_b - d_centroids[i].val[0]);
        b = (float) (p1_g - d_centroids[i].val[1]);
        c = (float) (p1_r - d_centroids[i].val[2]);
        temp = sqrt(a*a + b*b + c*c);
        if (temp < d) {
            d = temp;
            cluster = i;
        }
    }
    d_label[index] = cluster;
}

__global__ void GPU_labelToImage (uchar* d_label, Vec3b* d_centroids, unsigned char* d_image_final_B, unsigned char* d_image_final_G, unsigned char* d_image_final_R, int ROWS, int COLS, int CLUSTER_NUM) {
    int col = blockIdx.x;
    int row = threadIdx.x;
    int color = blockIdx.y;
    int index_image = row*COLS + col;
    int index_label = d_label[index_image];

    switch(color) {
      case 0 :
        d_image_final_B[index_image] = d_centroids[index_label].val[0];
        break;
      case 1:
        d_image_final_G[index_image] = d_centroids[index_label].val[1];
        break;
      case 2 :
        d_image_final_R[index_image] = d_centroids[index_label].val[2];
        break;
      default :
        printf("there is a bug lol\n" );
   }
}

// currently not updating centroids on GPU
// worth looking into: may not be faster due to atomic addition for averaging
__global__ void GPU_updateCentroid(Vec3b* d_centroids, Vec3i* d_centroids_new, int* d_count){
    int index = blockIdx.x;
    if (d_count[index] != 0) {
        d_centroids[index].val[0] = d_centroids_new[index].val[0]/d_count[index];
        d_centroids[index].val[1] = d_centroids_new[index].val[1]/d_count[index];
        d_centroids[index].val[2] = d_centroids_new[index].val[2]/d_count[index];
    }
    // resetting the values for next update
    d_count[index] = 0;
    // retardedly setting the bgr value to 0 one by one because cannot reference host methods
    d_centroids_new[index].val[0] = 0;
    d_centroids_new[index].val[1] = 0;
    d_centroids_new[index].val[2] = 0;
}

// generate a random RGB pixel as centroid
Vec3b randomPixel() {
    return Vec3b(rand()%255, rand()%255, rand()%255);
}

// same centroid updating fuction that is run on CPU
// average all the pixels that belong to one cluster and assign the average to be the new centroid
void updateCentroid(uchar* label, Vec3b* centroids, Mat original) {

    int cluster;
    // use Vec3i(vector3 int) to hold a bigger number
    Vec3i sum[CLUSTER_NUM];
    for (int i = 0; i < CLUSTER_NUM; i++) {
        sum[i] = Vec3i(0, 0, 0);
    }
    // dynamically allocating memory because CLUSTER_NUM changes at run time
    int* count = new int[CLUSTER_NUM];
    for (int i = 0; i < CLUSTER_NUM; i++) {
        count[i] = 0;
    }
    // the addition part of averaging
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            cluster = label[i*COLS + j];
            sum[cluster].val[0] += original.at<Vec3b>(i, j).val[0];
            sum[cluster].val[1] += original.at<Vec3b>(i, j).val[1];
            sum[cluster].val[2] += original.at<Vec3b>(i, j).val[2];
            count[cluster]++;
        }
    }
    // the division part of averaging
    for (int i = 0; i < CLUSTER_NUM; i++) {
        // randomize pixel if no pixels are assigned to the cluster
        if (count[i] == 0) {
            centroids[i] = randomPixel();
            continue;
        }
        sum[i].val[0] /= count[i];
        sum[i].val[1] /= count[i];
        sum[i].val[2] /= count[i];
        centroids[i] = sum[i];
    }

    // be careful of memory leaks!
    delete[] count;
}

int main(int argc, char** argv)
{
    // check for correct input format
    if (argc != 4) {
        printf("usage: ./gpu_fix filename(without '.jpg') #clusters #iterations\n");
        exit(-1);
    }

    // read inputs from commandline
    string fn = argv[1];
    CLUSTER_NUM = stoi(argv[2]);
    ITERATION = stoi(argv[3]);

    // check c++ version c++11 or c++98
    if( __cplusplus == 201103L ) std::cout << "C++11\n" ;
    else if( __cplusplus == 199711L ) std::cout << "C++98\n" ;
    else std::cout << "pre-standard C++\n" ;

    // print image information to commandline
    printf("number of clusters: %i\n", CLUSTER_NUM);
    printf("number of iterations: %i\n", ITERATION);
    printf("mode: GPU\n");
    
    // set random seed to a particular number to compare against CPU
    //srand(1);
    // randomize seed
    srand(time(NULL));
    Mat h_image, h_image_final;
    // use opencv to read the image
    h_image = imread(fn + ".jpg", IMREAD_COLOR);
    if (!h_image.data) {
        printf("No image data \n");
        return -1;
    }

    ROWS = h_image.rows;
    COLS = h_image.cols;

    printf("file size: %ix%i\n", COLS, ROWS);

    h_image_final.create(ROWS, COLS, CV_8UC1);

    // split original image to RGB channels
    Mat h_image_channel[3];
    split(h_image, h_image_channel);
    Mat h_image_final_channel[3];
    h_image_final_channel[0] = h_image_channel[0].clone();
    h_image_final_channel[1] = h_image_channel[1].clone();
    h_image_final_channel[2] = h_image_channel[2].clone();

    //generate random pixel for centroid

    Vec3b* h_centroids = (Vec3b*) malloc(CLUSTER_NUM * sizeof(Vec3b));
    for (int i = 0; i < CLUSTER_NUM; i++) {
        h_centroids[i] = randomPixel();
    }

    //generate a 2D array for labelling
    uchar* h_label = (uchar *) malloc(ROWS * COLS * sizeof(uchar));

    // gpu data structure
    // --original image-- three 1D array of unsigned char for BGR
    unsigned char* d_image_B;
    unsigned char* d_image_G;
    unsigned char* d_image_R;
    // dynamically allocating memory on the GPU
    hipMalloc((void**) &d_image_B, ROWS * COLS * sizeof(unsigned char));
    hipMalloc((void**) &d_image_G, ROWS * COLS * sizeof(unsigned char));
    hipMalloc((void**) &d_image_R, ROWS * COLS * sizeof(unsigned char));
    // copying data from CPU to GPU
    hipMemcpy(d_image_B, h_image_channel[0].data, ROWS * COLS * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_image_G, h_image_channel[1].data, ROWS * COLS * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_image_R, h_image_channel[2].data, ROWS * COLS * sizeof(unsigned char), hipMemcpyHostToDevice);

    // --new image-- three 1D array of unsigned char of BGR
    unsigned char* d_image_final_B;
    unsigned char* d_image_final_G;
    unsigned char* d_image_final_R;
    hipMalloc((void**) &d_image_final_B, ROWS * COLS * sizeof(unsigned char));
    hipMalloc((void**) &d_image_final_G, ROWS * COLS * sizeof(unsigned char));
    hipMalloc((void**) &d_image_final_R, ROWS * COLS * sizeof(unsigned char));


    // --labels--    1D array representing 2D array on GPU
    uchar* d_label;
    hipMalloc((void**) &d_label, ROWS * COLS * sizeof(uchar));

    // --centroids--   1D array of unsign char on GPU
    Vec3b* d_centroids;
    hipMalloc((void**) &d_centroids, CLUSTER_NUM * sizeof(Vec3b));

    // loop through different numbers of clusters and produce an image 
    CLUSTER_MAX = CLUSTER_NUM;
    for (int c = 1; c <= CLUSTER_MAX; c++) {
        CLUSTER_NUM = c;
        // randomize centroid pixels
        for (int i = 0; i < c; i++) {
            h_centroids[i] = randomPixel();
        }

        // the real k-means: for a given number of clusters, update the centroid k times and output an image
        for (int k = 0; k < ITERATION; k++) {

            // copy from host to device 
            hipMemcpy(d_centroids, h_centroids, CLUSTER_NUM * sizeof(Vec3b), hipMemcpyHostToDevice);

            // map all pixels to cluster
            GPU_mapToCluster<<<dim3(COLS), dim3(ROWS)>>>(d_label, d_centroids, d_image_B, d_image_G, d_image_R, ROWS, COLS, CLUSTER_NUM);
            //hipDeviceSynchronize();

            // update the image to the cluster colors
            GPU_labelToImage<<<dim3(COLS, 3), dim3(ROWS)>>>(d_label, d_centroids, d_image_final_B, d_image_final_G, d_image_final_R, ROWS, COLS, CLUSTER_NUM);
            //hipDeviceSynchronize();

            // download the calculated image channels from GPU to CPU
            hipMemcpy(h_image_final_channel[0].data, d_image_final_B, ROWS * COLS * sizeof(unsigned char), hipMemcpyDeviceToHost);
            hipMemcpy(h_image_final_channel[1].data, d_image_final_G, ROWS * COLS * sizeof(unsigned char), hipMemcpyDeviceToHost);
            hipMemcpy(h_image_final_channel[2].data, d_image_final_R, ROWS * COLS * sizeof(unsigned char), hipMemcpyDeviceToHost);

            //Merging the new channels into the calculated image
            merge(h_image_final_channel, 3, h_image_final);
            //imwrite(fn+"/" + fn + "_gpu_fix" + to_string(k) + ".jpg",h_image_final);

            // update the centroid locations
            // download the centroids and labels back to CPU
            hipMemcpy(h_centroids, d_centroids, CLUSTER_NUM * sizeof(Vec3b), hipMemcpyDeviceToHost);
            hipMemcpy(h_label, d_label, COLS * ROWS * sizeof(unsigned char), hipMemcpyDeviceToHost);

            updateCentroid(h_label, h_centroids, h_image);
        }

        
        putText(h_image_final, to_string(c), Point(20, 20), FONT_HERSHEY_COMPLEX_SMALL, 1, cvScalar(200,200,250), 1, CV_AA);
        // writes to file for each iteration of image
        imwrite(fn + "/" + fn + "_gpu_fix_" + to_string(c) + ".jpg", h_image_final);

    }
    imwrite(fn + "_gpu_time.jpg", h_image_final);
    return 0;
}

//display image
/*
namedWindow("Iteration " + to_string(k), WINDOW_AUTOSIZE );
if(!h_image_final.empty()){
    imshow("Iteration " + to_string(k), h_image_final);
}
waitKey(0);
*/